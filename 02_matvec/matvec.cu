#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// CUDA runtime
#include <hip/hip_runtime.h>

#define DTYPE float

const int maxThreadsPerBlock = 1024;

__device__ DTYPE mult(DTYPE *a, DTYPE *x, int size)
{
    int tid_x = threadIdx.x;

    int row = blockIdx.x;

    int bid_y = blockIdx.y;

    int bdim_x = blockDim.x;

    int col = tid_x + bid_y * bdim_x;

    return a[row * size + col] * x[col];
}

__device__ DTYPE reduce(thread_group g, DTYPE *cache, DTYPE val)
{
    int gtid_x = g.thread_rank();
    cache[gtid_x] = val;
    g.sync();
    for (int k = g.size() / 2; k > 0; k >>= 1)
    {
       
        if (gtid_x < k)
        {
            cache[gtid_x] += cache[gtid_x + k];
        }
        g.sync();
    }

    return cache[0];
}

__global__ void kernelAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
    __shared__ DTYPE cache[maxThreadsPerBlock];

    DTYPE val = mult(a, x, size);

    thread_group g = this_thread_block();
    auto tileIdx = g.thread_rank() / 32;
    DTYPE* t = &cache[32 * tileIdx];

    thread_group tile = tiled_partition(g, 32);

    DTYPE sum = reduce(tile, t, val);

    //printf("%f \n", cache[0]);

    if (tile.thread_rank() == 0) atomicAdd(&y[blockIdx.x], sum);

    //if (row < size && col < size) atomicAdd(&y[row], a[row * size + col] * x[col]);
}



//A mit Werten füllen (hier einfach 1en)
void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++)
      x[i]= (DTYPE)(i+1);
}

void hostAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
    //TODO: Hier soll der Host A*x=y berechnen
    for (unsigned int i = 0; i < size; i++)
    {
        y[i] = 0;
        for (unsigned int j = 0; j < size; j++)
        {
            y[i] += a[j * size + i] * x[j];
        }
    }
}




bool checkResult(DTYPE *yh, DTYPE *yd, int size)
{
   bool res=true;
   for (int i=0;i<size;i++)
   {
      res&=(yh[i]==yd[i]);
      if (i<10) printf("%f %f\n",yh[i],yd[i]);
   }
   return res;
}

/*
   Main Routine: 
   Input: i,[threads]
   Berechnet A*x=y auf der GPU wobei A eine Größe von R^{n x n} hat, mit
   n=1024*i
*/
int main(int argc, char**argv)
{
    int i = 1;
    int t = 512;
    // if (argc>1)
    // {
    //    i=atoi(argv[1]);
    //    if (argc>2) t=atoi(argv[2]);
    // }
    // else 
    // {
    //    printf("Usage: %s i [threads] \n",argv[0]);
    //    return -1;
    // }
    // printf("size %i \n", i);
    int size = 1024 * i;
    //Datenfelder anlegen für Host
    DTYPE *a_host, *yd_host, *yh_host, *x_host;
    //und Device
    DTYPE *a_dev, *y_dev, *x_dev;
    //Events für die Zeitmessung
    hipEvent_t start, end;
    //Zeiten: 
    //htd: Host->Device Memcpy von A und x
    float htd_time = 0.0;
    //dth: Device->Host Memcpy von y
    float dth_time = 0.0;
    //kernelA, kernelAT
    float kernelA_time = 0.0;
    float kernelAT_time = 0.0;

    //TODO: Host Speicher anlegen und A und x füllen
    a_host = (DTYPE*)malloc(size * size * sizeof(DTYPE));
    x_host = (DTYPE*)malloc(size * sizeof(DTYPE));
    yd_host = (DTYPE*)malloc(size * sizeof(DTYPE));
    yh_host = (DTYPE*)malloc(size * sizeof(DTYPE));

    fillA(a_host, size);
    fillX(x_host, size);

    //TODO: CUDA Events erstellen
    hipEventCreate(&start);
    hipEventCreate(&end);

    //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
    hipMalloc((void**)&a_dev, size*size * sizeof(DTYPE));
    hipMalloc((void**)&x_dev, size * sizeof(DTYPE));
    hipMalloc((void**)&y_dev, size * sizeof(DTYPE));

    //TODO: Host->Device Memcpy von A und x + Zeitmessung
    hipEventRecord(start, 0);
    hipMemcpy(x_dev, x_host, size * sizeof(DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(a_dev, a_host, size*size * sizeof(DTYPE), hipMemcpyHostToDevice);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipEventElapsedTime(&htd_time, start, end);

    //Konfiguration der CUDA Kernels
    dim3 threads(512);
    dim3 grid(size, size / threads.x);

    //TODO: kernelAx ausführen und Zeit messen
    hipEventRecord(start, 0);
    kernelAx<<<grid, threads>>>(a_dev, x_dev, y_dev, size);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipEventElapsedTime(&kernelA_time, start, end);

    //TODO: Device->Host Memcpy für y_dev -> yd_host
    hipEventRecord(start, 0);
    hipMemcpy(yd_host, y_dev, size * sizeof(DTYPE), hipMemcpyDeviceToHost);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipEventElapsedTime(&dth_time, start, end);

    printf("GPU timing in ms: h->d: %f kernelAx: %f kernelATx: %f d->h: %f\n", htd_time, kernelA_time, kernelAT_time, dth_time);

    hostAx(a_host, x_host, yh_host, size);
    //TODO: checkResult aufrufen
    checkResult(yh_host, yd_host, size);

    //TODO: Speicher freigeben (Host UND Device)
    hipFree(a_dev);
    hipFree(x_dev);
    hipFree(y_dev);

    free(a_host);
    free(x_host);
    free(yh_host);
    free(yd_host);
}
